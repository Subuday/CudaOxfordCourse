#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

__global__ void init(float *x) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    x[tid] = (float)threadIdx.x;
}

__global__ void add(float *x, float *y, float *z) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  z[tid] = x[tid] + y[tid];
}

int main(int argc, const char **argv) {
    float *h_x, *d_x1, *d_x2, *d_x3;
    int nblocks, nthreads, nsize, n;

    findCudaDevice(argc, argv);

    nblocks = 2;
    nthreads = 4;
    nsize = nblocks * nthreads;

    h_x = (float *)malloc(nsize  * sizeof(float));
    checkCudaErrors(hipMalloc((void **)&d_x1, nsize * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x2, nsize * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x3, nsize * sizeof(float)));

    init<<<nblocks, nthreads>>>(d_x1);
    getLastCudaError("init execution failed\n");
    init<<<nblocks, nthreads>>>(d_x2);
    getLastCudaError("init execution failed\n");
    add<<<nblocks, nthreads>>>(d_x1, d_x2, d_x3);
    getLastCudaError("add execution failed\n");

    checkCudaErrors(hipMemcpy(h_x, d_x3, nsize * sizeof(float), hipMemcpyDeviceToHost));
    for (n = 0; n < nsize; n++) printf(" n,  x  =  %d  %f \n", n, h_x[n]);

    // free memory
    checkCudaErrors(hipFree(d_x1));
    checkCudaErrors(hipFree(d_x2));
    checkCudaErrors(hipFree(d_x3));
    free(h_x);

    hipDeviceReset();

    return 0;
}