#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  int deviceId = 0;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, deviceId);

  int maxBlocksPerSM;
  int maxThreadsPerBlock;
  int maxThreadsPerSM;

  hipDeviceGetAttribute(&maxBlocksPerSM, hipDeviceAttributeMaxBlocksPerMultiprocessor,
                         deviceId);

  hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock,
                         deviceId);

  maxThreadsPerSM = maxBlocksPerSM * maxThreadsPerBlock;

  std::cout << "Max Blocks Per SM: " << maxBlocksPerSM << std::endl;
  std::cout << "Max Threads Per Block: " << maxThreadsPerBlock << std::endl;
  std::cout << "Max Threads Per SM (nsize): " << maxThreadsPerSM << std::endl;

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, deviceId);
  std::cout << "Device name: " << prop.name << std::endl;
  std::cout << "Compute capability: " << prop.major << "." << prop.minor
            << std::endl;

  return 0;
}